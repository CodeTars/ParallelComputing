#include "hip/hip_runtime.h"
#include "h_gpu.h"

#define Swap(a, b)    \
    {                 \
        double t = a; \
        a = b;        \
        b = t;        \
    }

/**
 * 单线程，从[Ak,k ... An-1,k]中寻找列主元的索引l,
 * 后交换k,l两行 [Ak,k ... Ak,n]<-->[Al,k ... Al,n]
 * */
__global__ void SwapRow(double* a, int n, int k)
{
    /* 求第k列的最大元 */
    int l = k;
    double mx = fabs(a[(n + 1) * k + k]);
    for (int i = k + 1; i < n; i++) {
        if (fabs(a[(n + 1) * i + k]) > mx) {
            mx = fabs(a[(n + 1) * i + k]);
            l = i;
        }
    }
    /* 交换k,l两行 */
    if (k != l) {
        for (int j = k; j < n + 1; j++) {
            Swap(a[(n + 1) * k + j], a[(n + 1) * l + j]);
        }
    }
}

/**
 * 多线程, 并行消元
 * 需要操作的元素为
 * ----> x
 * | |Ak+1,k+1 ... Ak+1,n|
 * ' |                   |
 * y |    .           .  |
 *   |    .           .  |
 *   |An-1,k+1 ... An-1,n|
 * 
 * ceil(n / EliminationBlockDim_Y)(gridDim.y) * ceil((n + 1) / EliminationBlockDim_X)(gridDim.x) blocks per Grid
 * EliminationBlockDim_Y(blockDim.y) * EliminationBlockDim_X(blockDim.x) threads per Block
 * */
#define EliminationBlockDim_Y 32
#define EliminationBlockDim_X 32
__global__ void GaussElimination(double* a, int n, int k)
{
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    if (i > k && i < n && j > k && j < n + 1) {
        a[(n + 1) * i + j] -= (a[(n + 1) * i + k] / a[(n + 1) * k + k]) * a[(n + 1) * k + j];
    } // 消元
}

int main(int argc, char* argv[])
{
    clock_t tic, toc;

    /* 方程组（增广矩阵）输入 */
    int n;
    double* A; // 主机内存中的增广矩阵
    Input(A, n);

    /**
     * 消元过程线程划分
     * GRID_DIM.x = 矩阵的列数n+1被分成多少块, GRID_DIM.y = 矩阵的行数n被分成多少块
     * */
    const dim3 GRID_DIM(ceil(1.0 * (n + 1) / EliminationBlockDim_X), ceil(1.0 * n / EliminationBlockDim_Y), 1);
    const dim3 BLOCK_DIM(EliminationBlockDim_X, EliminationBlockDim_Y, 1);

    /* 内存分配 */
    tic = clock();
    double* a = NULL; // GPU内存中的增广矩阵
    hipMalloc((void**)&a, sizeof(double) * n * (n + 1));
    toc = clock();
    double malloc_time = double(toc - tic) / CLOCKS_PER_SEC;

    /* CPU->GPU */
    tic = clock();
    hipMemcpy(a, A, sizeof(double) * n * (n + 1), hipMemcpyHostToDevice);
    toc = clock();
    double copy_time = double(toc - tic) / CLOCKS_PER_SEC;

    /* 消元过程 */
    GpuTimer timer;
    float kernel_time1 = 0, kernel_time2 = 0, kernel_time3 = 0;
    for (int k = 0; k < n - 1; k++) {

        /* 交换两行 */
        timer.Start();
        SwapRow<<<1, 1>>>(a, n, k);
        timer.Stop();
        kernel_time2 += timer.Elapsed() / 1000.0;

        /* 高斯消元 */
        timer.Start();
        GaussElimination<<<GRID_DIM, BLOCK_DIM>>>(a, n, k);
        timer.Stop();
        kernel_time3 += timer.Elapsed() / 1000.0;

    } // 此时增广矩阵已变换成了上三角阵

    /* GPU->CPU */
    tic = clock();
    hipMemcpy(A, a, sizeof(double) * n * (n + 1), hipMemcpyDeviceToHost);
    toc = clock();
    copy_time += double(toc - tic) / CLOCKS_PER_SEC;

    /* cpu内回代过程 */
    tic = clock();
    backSubstitution(A, n);
    toc = clock();
    double backSub_time = double(toc - tic) / CLOCKS_PER_SEC;

    /* 输出 */
    Output(A, n);
    PrintTime("gauss1", n, malloc_time, copy_time, kernel_time1, kernel_time2, kernel_time3, backSub_time);

    delete[] A;
    hipFree(a);
    return 0;
}
